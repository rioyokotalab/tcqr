#include "hip/hip_runtime.h"
#include <type_traits>
#include <mma.h>
#include <hip/hip_fp16.h>
#include <cutf/type.hpp>
#include <cutf/math.hpp>
#include "tcqr.hpp"
#include "utils.hpp"

//#define DEBUG

namespace{
constexpr std::size_t warp_size = 32; // 本当はwarpSizeを使いたい
constexpr unsigned fragment_dimension = 16;

template <class Func>
__device__ void debug_func(unsigned warp_id,	Func run_func){
#ifdef DEBUG
	if(warp_id == 0){
		run_func();
	}
#endif
}

// 2乗和
// 内部計算をS型で行い，S型で返す
// sum(ptr[start_id] : ptr[15])
template <class T, class S>
__device__ S get_norm2_16(T* const ptr, const std::size_t size, unsigned warp_id){
	auto tmp = cutf::cuda::type::cast<S>(0.0f);
	
	// load
	if(warp_id < size){
		tmp = cutf::cuda::type::cast<S>(ptr[warp_id]);
		tmp = tmp * tmp;
	}

	// shfl allreduce
	for(auto mask = (warp_size>>1); mask > 0; mask >>= 1){
		tmp += __shfl_xor_sync(0xffffffff, tmp, mask);
	}
	return tmp;
}

// 結合アクセセスを意識
template <class T, class S>
__device__ void copy_16x16(T* const dest_ptr, const S* const src_ptr, unsigned warp_id){
#pragma unroll 
	for(unsigned i = 0; i < fragment_dimension * fragment_dimension / warp_size; i++){
		dest_ptr[warp_size * i + warp_id] = cutf::cuda::type::cast<T>(src_ptr[warp_size * i + warp_id]);
	}
}
template <class T, class S>
__device__ void copy_16x16(T* const dest_ptr, const S* const src_ptr, std::size_t m, std::size_t n, unsigned warp_id){
#pragma unroll 
	for(unsigned i = 0; i < fragment_dimension * fragment_dimension / warp_size; i++){
		const auto index = warp_size * i + warp_id;
		const auto x = index / fragment_dimension;
		const auto y = index % fragment_dimension;
		auto val = cutf::cuda::type::cast<S>(0.0f);
		if(x < n && y < m)
			val = cutf::cuda::type::cast<S>(src_ptr[x * m + y]);;

		dest_ptr[index] = val;
	}
}
// TODO : 結合アクセス
template <class T, class S>
__device__ void copy_16x16(T* const dest_ptr, std::size_t m, std::size_t n, const S* const src_ptr, unsigned warp_id){
#pragma unroll 
	for(unsigned i = 0; i < fragment_dimension * fragment_dimension / warp_size; i++){
		const auto index = warp_size * i + warp_id;
		const auto x = index / fragment_dimension;
		const auto y = index % fragment_dimension;
		if(x < n && y < m)
			dest_ptr[x * m + y] = cutf::cuda::type::cast<S>(src_ptr[index]);
	}
}

// Globalメモリアクセスを結合アクセスにすると遅くなる気がする．
// 要素位置(x, y)の計算などで差がつくのかも?
template <class T, class S>
__device__ void copy_16x16_T(T* const dest_ptr, std::size_t m, std::size_t n, const S* const src_ptr, unsigned warp_id){
#pragma unroll
	for(unsigned i = 0; i < fragment_dimension * fragment_dimension / warp_size; i++){
		const auto index = warp_size * i + warp_id;
		const auto x = index / fragment_dimension;
		const auto y = index % fragment_dimension;
		if(x < n && y < m)
			dest_ptr[m * y + x] = cutf::cuda::type::cast<S>(src_ptr[index]);
	}
}

template <class T, class S>
__device__ void copy_16(T* const dest_ptr, const S* const src_ptr, unsigned warp_id){
	if(warp_id < fragment_dimension){
		dest_ptr[warp_id] = cutf::cuda::type::cast<T>(src_ptr[warp_id]);
	}
}

// 行列積
// Bが対称行列の場合，C <- A * BはC <- A^T * Bと同値
// 連続メモリアクセスのためTNで計算する
template <class T>
__device__ void matmul_16x16_TN(T* const c, const T* const a, const T* const b, unsigned warp_id){
	/* 行列Cを1ワープで計算する
	 * スレッドによる分割方法は
	 * C(列優先) = 
	 * -------------------- -
	 * |   |   | ... |    | ^
	 * | 0 | 2 | ... | 30 | |
	 * |   |   | ... |    | |
	 * -------------------- 16
	 * |   |   | ... |    | |
	 * | 1 | 3 | ... | 31 | |
	 * |   |   | ... |    | v
	 * -------------------- -
	 * <--------16-------->
	 * の様に分割する．
	 * (start_i, j)は各スレッドの書き込み先の
	 * 先頭の要素
	 */
	// (x % 2) <-> (x & 0x1)
	const auto start_i = (warp_id & 0x1) * (fragment_dimension/2);
	// (x / 2) <-> (x >> 1)
	const auto j = (warp_id >> 1);
	T sums[fragment_dimension/2];

	for(std::size_t i = start_i; i < fragment_dimension / 2 + start_i; i++){
		auto sum = cutf::cuda::type::cast<T>(0.0f);
		for(std::size_t k = 0; k < fragment_dimension; k++){
			sum += a[fragment_dimension * i + k] * b[fragment_dimension * j + k];
		}
		sums[i - start_i] = sum;
	}
	__syncthreads();

	// 一度バッファ(レジスタ)に貯めてからメモリに書き込み
	for(std::size_t i = start_i; i < fragment_dimension / 2 + start_i; i++){
		c[fragment_dimension * j + i] = sums[i - start_i];
	}
}

template <class T>
__device__ void make_identity_matrix(T* const dest_ptr, std::size_t m, unsigned warp_id){
	for(unsigned i = 0; i < fragment_dimension * fragment_dimension / warp_size; i++){
		const auto index = warp_size * i + warp_id;
		if(index % (fragment_dimension + 1) == 0) dest_ptr[index] = cutf::cuda::type::cast<T>(1.0f);
		else dest_ptr[index] = cutf::cuda::type::cast<T>(0.0f);
	}
}

// 結合アクセセスを意識
template <class T, class S>
__device__ void make_h(T* const h, const S* const u, const S norm_u2, unsigned warp_id){
#pragma unroll 
	for(unsigned i = 0; i < fragment_dimension * fragment_dimension / warp_size; i++){
		const auto index = warp_size * i + warp_id;
		const auto x = index / fragment_dimension;
		const auto y = index % fragment_dimension;

		// 単位行列生成は make_identity_matrix関数を用いない
		// メモリアクセスを減らせる
		T val;
		if(index % (fragment_dimension + 1) == 0) val = cutf::cuda::type::cast<T>(1.0f);
		else val = cutf::cuda::type::cast<T>(0.0f);

		val -= cutf::cuda::type::cast<T>(2.0f) * cutf::cuda::type::cast<T>(u[x] * u[y] * cutf::cuda::math::rcp(norm_u2));
		h[index] = val;
	}
}

// Q,R の更新
template <class Input_t, class Output_t>
__device__ void update_qr_tc(
		Output_t* const out_q, 
		Output_t* const out_r, 
		const Input_t* const in_q, 
		const Input_t* const in_r, 
		const Input_t* const in_h){
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 700
	nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, fragment_dimension, fragment_dimension, fragment_dimension, half, nvcuda::wmma::col_major> in_h_fragment;
	nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, fragment_dimension, fragment_dimension, fragment_dimension, half, nvcuda::wmma::col_major> in_q_fragment;
	nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, fragment_dimension, fragment_dimension, fragment_dimension, half, nvcuda::wmma::col_major> in_r_fragment;
	nvcuda::wmma::fragment<nvcuda::wmma::accumulator, fragment_dimension, fragment_dimension, fragment_dimension, Output_t> out_q_fragment;
	nvcuda::wmma::fragment<nvcuda::wmma::accumulator, fragment_dimension, fragment_dimension, fragment_dimension, Output_t> out_r_fragment;

	nvcuda::wmma::fill_fragment(out_q_fragment, cutf::cuda::type::cast<Output_t>(0.0f));
	nvcuda::wmma::fill_fragment(out_r_fragment, cutf::cuda::type::cast<Output_t>(0.0f));

	nvcuda::wmma::load_matrix_sync(in_h_fragment, in_h, fragment_dimension);
	nvcuda::wmma::load_matrix_sync(in_q_fragment, in_q, fragment_dimension);
	nvcuda::wmma::load_matrix_sync(in_r_fragment, in_r, fragment_dimension);

	nvcuda::wmma::mma_sync(out_q_fragment, in_h_fragment, in_q_fragment, out_q_fragment);
	nvcuda::wmma::mma_sync(out_r_fragment, in_h_fragment, in_r_fragment, out_r_fragment);

	nvcuda::wmma::store_matrix_sync(out_q, out_q_fragment, fragment_dimension, nvcuda::wmma::mem_col_major);
	nvcuda::wmma::store_matrix_sync(out_r, out_r_fragment, fragment_dimension, nvcuda::wmma::mem_col_major);
#endif
}

// 非TCQ,R更新関数
template <class T, bool UseTC>
__device__ void update_qr(T* const out_q, T* const out_r, const T* const in_q, const T* const in_r, const T* const in_h,unsigned warp_id){
	// TODO : hの再利用
	matmul_16x16_TN(out_q, in_h, in_q, warp_id);
	matmul_16x16_TN(out_r, in_h, in_r, warp_id);
}
template <>
__device__ void update_qr<half, true>(half* const out_q, half* const out_r, const half* const in_q, const half* const in_r, const half* const in_h,unsigned warp_id){
	update_qr_tc<half, half>(out_q, out_r, in_q, in_r, in_h);
}

// tcqr
// 入出力はShared memoryで
// out_q/out_rの初期化は関数の手前で行っておくこと
// out_q <- Identity matrix
// out_r <- a
template <class T, class Norm_t, bool UseTC>
__device__ void qr16x16_core(T* const out_q, T* const out_r, 
		T* const h, T* const u,
		const std::size_t m, const std::size_t n, unsigned warp_id){
	for(std::size_t k = 0; k < n; k++){
		debug_func(warp_id,
				[&k](){printf(
					"//---------------------\n"
					"// k = %lu\n"
					"//---------------------\n"
					, k);});
		debug_func(warp_id,
				[&m, &n, &out_r](){utils::print_matrix(out_r, 16, 16, "r");});
		debug_func(warp_id,
				[&m, &out_q](){utils::print_matrix(out_q, 16, 16, "q");});

		copy_16(u, out_r + fragment_dimension * k, warp_id);
		if(warp_id < k){
			u[warp_id] = cutf::cuda::type::cast<T>(0.0f);
		}
		debug_func(warp_id,
				[&u](){utils::print_matrix(u, 1, 16, "u");});

		const auto norm_u = cutf::cuda::math::sqrt(cutf::cuda::type::cast<T>(get_norm2_16<T, Norm_t>(u, m, warp_id)));
		if(warp_id == k){
			u[warp_id] += norm_u * cutf::cuda::math::sign(u[warp_id]);
		}
		debug_func(warp_id,
				[&u](){utils::print_matrix(u, 1, 16, "u+");});

		const auto norm_u2 = cutf::cuda::type::cast<T>(get_norm2_16<T, Norm_t>(u, m, warp_id));
		make_h(h, u, norm_u2, warp_id);
		update_qr<T, UseTC>(out_q, out_r, out_q, out_r, h, warp_id);
	}
}
__device__ void qr16x16_f32tc_core(
		float * const q_f32, float* const r_f32,
		half* const q_f16, half* const r_f16,
		float* const u_f32, half* const h_f16,
		const std::size_t m, const std::size_t n, unsigned warp_id){
	for(std::size_t k = 0; k < n; k++){
		debug_func(warp_id,
				[&k](){printf(
					"//---------------------\n"
					"// k = %lu\n"
					"//---------------------\n"
					, k);});
		debug_func(warp_id,
				[&q_f32](){utils::print_matrix(q_f32, 16, 16, "q");});
		debug_func(warp_id,
				[&r_f32](){utils::print_matrix(r_f32, 16, 16, "r");});

		copy_16(u_f32, r_f32 + fragment_dimension * k, warp_id);
		if(warp_id < k){
			u_f32[warp_id] = 0.0f;
		}
		debug_func(warp_id,
				[&u_f32](){utils::print_matrix(u_f32, 1, 16, "u");});

		const auto norm_u = cutf::cuda::math::sqrt(get_norm2_16<float, float>(u_f32, m, warp_id));
		if(warp_id == k){
			u_f32[warp_id] += norm_u * cutf::cuda::math::sign(u_f32[warp_id]);
		}
		debug_func(warp_id,
				[&u_f32](){utils::print_matrix(u_f32, 1, 16, "u+");});

		const auto norm_u2 = get_norm2_16<float, float>(u_f32, m, warp_id);
		make_h(h_f16, u_f32, norm_u2, warp_id);
		// q,r の型変換
		copy_16x16<half, float>(q_f16, q_f32, warp_id);
		copy_16x16<half, float>(r_f16, r_f32, warp_id);

		update_qr_tc<half, float>(q_f32, r_f32, q_f16, r_f16, h_f16);
	}
}

// kernel
template <class T, class Norm_t, bool UseTC>
__global__ void qr16x16_kernel(T* const q, T* const r, const T* const a, const std::size_t m, const std::size_t n){
	// (x % 32) <-> (x & 0x1f)
	const auto warp_id = threadIdx.x & 0x1f;
	__shared__ T q_shared[fragment_dimension * fragment_dimension];
	__shared__ T r_shared[fragment_dimension * fragment_dimension];
	__shared__ T h[fragment_dimension * fragment_dimension];
	__shared__ T u[fragment_dimension];


	copy_16x16<T, T>(r_shared, a, m, n, warp_id);
	make_identity_matrix(q_shared, m, warp_id);

	qr16x16_core<T, Norm_t, UseTC>(q_shared, r_shared,
			h, u,
		   	m, n, warp_id);

	copy_16x16<T, T>(r, m, n, r_shared, warp_id);
	copy_16x16_T<T, T>(q, m, m, q_shared, warp_id);
}

// 単精度入出力TC使用
__global__ void qr16x16_f32tc_kernel(float* const q, float* const r, const float* const a, const std::size_t m, const std::size_t n){
	// (x % 32) <-> (x & 0x1f)
	const auto warp_id = threadIdx.x & 0x1f;
	__shared__ float q_shared_f32[fragment_dimension * fragment_dimension];
	__shared__ float r_shared_f32[fragment_dimension * fragment_dimension];
	__shared__ half q_shared_f16[fragment_dimension * fragment_dimension];
	__shared__ half r_shared_f16[fragment_dimension * fragment_dimension];
	__shared__ half h_shared[fragment_dimension * fragment_dimension];
	__shared__ float u_shared[fragment_dimension];

	copy_16x16(r_shared_f32, a, m, n, warp_id);
	make_identity_matrix(q_shared_f32, m, warp_id);

	qr16x16_f32tc_core(q_shared_f32, r_shared_f32,
			q_shared_f16, r_shared_f16,
			u_shared, h_shared,
			m, n, warp_id);


	copy_16x16(r, m, n, r_shared_f32, warp_id);
	copy_16x16_T(q, m, m, q_shared_f32, warp_id);
}
} // noname namespace

// if constexpr が使えるようになったら書き直せ!!!!
template <class Input_t, class Output_t, class Norm_t, bool UseTC>
void tcqr::qr16x16(Output_t *const q, Output_t *const r, const Input_t *const a, const std::size_t m, const std::size_t n){
	qr16x16_kernel<Output_t, Norm_t, UseTC><<<1, warp_size>>>(q, r, a, m, n);
}
template <> void tcqr::qr16x16<float, float, float, true>(float *const q, float *const r, const float *const a, const std::size_t m, const std::size_t n){qr16x16_f32tc_kernel<<<1, warp_size>>>(q, r, a, m, n);};
template void tcqr::qr16x16<half, half, half, true>(half *const, half *const, const half *const, const std::size_t, const std::size_t);
template void tcqr::qr16x16<half, half, float, true>(half *const, half *const, const half *const, const std::size_t, const std::size_t);
template void tcqr::qr16x16<half, half, half, false>(half *const, half *const, const half *const, const std::size_t, const std::size_t);
template void tcqr::qr16x16<half, half, float, false>(half *const, half *const, const half *const, const std::size_t, const std::size_t);
template void tcqr::qr16x16<float, float, float, false>(float *const, float *const, const float *const, const std::size_t, const std::size_t);
//template <> void tcqr::qr16x16<double, double, double, false>(double *const q, double *const r, const double *const a, const std::size_t m, const std::size_t n){qr16x16_heterogeneous_kernel<double, double, double, true><<<1, warp_size>>>(q, r, a, m, n);};
