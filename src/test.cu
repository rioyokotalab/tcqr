#include "hip/hip_runtime.h"
#include <random>
#include <iostream>
#include <cutf/cublas.hpp>
#include <cutf/type.hpp>
#include <cutf/memory.hpp>
#include "test.hpp"
#include "utils.hpp"
#include "tcqr.hpp"

// #define PRINT_MATRIX

namespace{
template <class T>std::string get_type_name();
template <> std::string get_type_name<float>(){return "float";};
template <> std::string get_type_name<half>(){return "half";};

// 副作用があるっぽく見せるために適当なポインタ引数を取るようにする
// nvccの最適化で消されないようにするため
__global__ void tc_warning_kernel(void* p){
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ < 700
	printf("This device cannot execute code using TensorCore\n");
#endif
}
void tc_warning(){
	tc_warning_kernel<<<1, 1>>>(nullptr);
}
}

template <class T, class Norm_t, bool UseTC, std::size_t test_count>
void test::time::qr(const std::size_t m, const std::size_t n, const float* const a){
	if(UseTC)
		tc_warning();
	auto d_matrix_a = cutf::cuda::memory::get_device_unique_ptr<T>(m * n);
	auto d_matrix_r = cutf::cuda::memory::get_device_unique_ptr<T>(m * n);
	auto d_matrix_q = cutf::cuda::memory::get_device_unique_ptr<T>(m * m);
	auto d_matrix_qr = cutf::cuda::memory::get_device_unique_ptr<float>(m * n);

	auto h_matrix_a = cutf::cuda::memory::get_host_unique_ptr<T>(m * n);
	auto h_matrix_r = cutf::cuda::memory::get_host_unique_ptr<T>(m * n);
	auto h_matrix_q = cutf::cuda::memory::get_host_unique_ptr<T>(m * m);
	auto h_matrix_qr = cutf::cuda::memory::get_host_unique_ptr<float>(m * n);

	// print type information{{{
	utils::print_value(test_count, "Test count");
	utils::print_value(std::to_string(m) + " x " + std::to_string(n), "Matrix size");
	utils::print_value(get_type_name<T>(), "Input/Output type");
	utils::print_value(get_type_name<Norm_t>(), "Norm type");
	utils::print_value((UseTC ? "true" : "false"), "Use TC?");
	// }}}

	// copy
	for(std::size_t i = 0; i < m * n; i++){
		h_matrix_a.get()[i] = cutf::cuda::type::cast<Input_t>(a[i]);
	}

	cutf::cuda::memory::copy(d_matrix_a.get(), h_matrix_a.get(), m * n);
	auto elapsed_time = utils::get_elapsed_time(
			[&d_matrix_q, &d_matrix_r, &d_matrix_a, &m, &n](){
			for(std::size_t c = 0; c < test_count; c++)
				tcqr::qr16x16<T, Norm_t, UseTC>(d_matrix_q.get(), d_matrix_r.get(), d_matrix_a.get(), m, n);
			hipDeviceSynchronize();
			});
	utils::print_value(elapsed_time / test_count, "Elapsed time", "ms");
	utils::print_value(test_count * 16 * 16 * 16 * 2 * 2 * (n-1) / elapsed_time * 1000.0 / 1000000000.0, "", "GFLOPS");


	// 検証
	Output_t one = cutf::cuda::type::cast<Output_t>(1.0f);
	Output_t zero = cutf::cuda::type::cast<Output_t>(0.0f);
	auto cublas = cutf::cublas::get_cublas_unique_ptr();
	cutf::cublas::gemm(
			*cublas.get(),
			HIPBLAS_OP_N, HIPBLAS_OP_N,
			m, n, m,
			&one,
			d_matrix_q.get(), m,
			d_matrix_r.get(), m,
			&zero,
			d_matrix_qr.get(), m
			);
	cutf::cuda::memory::copy(h_matrix_qr.get(), d_matrix_qr.get(), m * n);

#ifdef PRINT_MATRIX
	cutf::cuda::memory::copy(h_matrix_q.get(), d_matrix_q.get(), m * m);
	cutf::cuda::memory::copy(h_matrix_r.get(), d_matrix_r.get(), m * n);
	utils::print_matrix(h_matrix_a.get(), m, n, std::string("A").c_str());
	std::cout<<std::endl;
	utils::print_matrix(h_matrix_q.get(), m, m, std::string("Q").c_str());
	std::cout<<std::endl;
	utils::print_matrix(h_matrix_r.get(), m, n, std::string("R").c_str());
	std::cout<<std::endl;
	utils::print_matrix(h_matrix_qr.get(), m, n, std::string("QR").c_str());
#endif

	const auto error = utils::get_error(a, h_matrix_qr.get(), m, n);
	utils::print_value(error , "error");
	std::cout<<std::endl;
}

template <class T, class Norm_t, bool UseTC, std::size_t test_count>
void test::precision::qr(const std::size_t m, const std::size_t n){
	if(UseTC)
		tc_warning();
	auto d_matrix_a = cutf::cuda::memory::get_device_unique_ptr<T>(m * n);
	auto d_matrix_r = cutf::cuda::memory::get_device_unique_ptr<T>(m * n);
	auto d_matrix_q = cutf::cuda::memory::get_device_unique_ptr<T>(m * m);
	auto d_matrix_qr = cutf::cuda::memory::get_device_unique_ptr<float>(m * n);

	auto h_matrix_a = cutf::cuda::memory::get_host_unique_ptr<T>(m * n);
	auto h_matrix_a_f32 = cutf::cuda::memory::get_host_unique_ptr<float>(m * n);
	auto h_matrix_r = cutf::cuda::memory::get_host_unique_ptr<T>(m * n);
	auto h_matrix_q = cutf::cuda::memory::get_host_unique_ptr<T>(m * m);
	auto h_matrix_qr = cutf::cuda::memory::get_host_unique_ptr<float>(m * n);

	// print type information{{{
	utils::print_value(test_count, "Test count");
	utils::print_value(std::to_string(m) + " x " + std::to_string(n), "Matrix size");
	utils::print_value(get_type_name<T>(), "Input/Output type");
	utils::print_value(get_type_name<Norm_t>(), "Norm type");
	utils::print_value((UseTC ? "true" : "false"), "Use TC?");
	// }}}

	std::mt19937 mt(std::random_device{}());
	std::uniform_real_distribution<float> dist(-1.0f, 1.0f);
	float error_sum = 0.0f;
	for(std::size_t i = 0; i < test_count; i++){
		// copy
		for(std::size_t i = 0; i < m * n; i++){
			h_matrix_a.get()[i] = cutf::cuda::type::cast<Input_t>(dist(mt));
		}

		cutf::cuda::memory::copy(d_matrix_a.get(), h_matrix_a.get(), m * n);

		// 検証
		Output_t one = cutf::cuda::type::cast<Output_t>(1.0f);
		Output_t zero = cutf::cuda::type::cast<Output_t>(0.0f);
		auto cublas = cutf::cublas::get_cublas_unique_ptr();
		tcqr::qr16x16<Input_t, Output_t, Norm_t, UseTC>(d_matrix_q.get(), d_matrix_r.get(), d_matrix_a.get(), m, n);
		cutf::cublas::gemm(
				*cublas.get(),
				HIPBLAS_OP_N, HIPBLAS_OP_N,
				m, n, m,
				&one,
				d_matrix_q.get(), m,
				d_matrix_r.get(), m,
				&zero,
				d_matrix_qr.get(), m
				);
		cutf::cuda::memory::copy(h_matrix_qr.get(), d_matrix_qr.get(), m * n);
		const auto error = utils::get_error(h_matrix_a.get(), h_matrix_qr.get(), m, n);
		error_sum += error;
	}
	utils::print_value(error_sum/test_count , "error avg");
	std::cout<<std::endl;
}

template void test::time::qr<half, half, true>(const std::size_t, const std::size_t, const float* const);
template void test::time::qr<half, half, false>(const std::size_t, const std::size_t, const float* const);
template void test::time::qr<half, float, true>(const std::size_t, const std::size_t, const float* const);
template void test::time::qr<half, float, false>(const std::size_t, const std::size_t, const float* const);
template void test::time::qr<float, float, false>(const std::size_t, const std::size_t, const float* const);
template void test::time::qr<float, float, true>(const std::size_t, const std::size_t, const float* const);

template void test::precision::qr<half, half, true>(const std::size_t, const std::size_t);
template void test::precision::qr<half, half, false>(const std::size_t, const std::size_t);
template void test::precision::qr<half, float, true>(const std::size_t, const std::size_t);
template void test::precision::qr<half, float, false>(const std::size_t, const std::size_t);
template void test::precision::qr<float, float, false>(const std::size_t, const std::size_t);
template void test::precision::qr<float, float, true>(const std::size_t, const std::size_t);

template <class T, class Norm_t, bool UseTC, std::size_t test_count>
void test::time::eigen(const std::size_t n, const float* const a){
	//eigen_eigen(a, n);return;
	if(UseTC)
		tc_warning();
	auto d_matrix_a = cutf::cuda::memory::get_device_unique_ptr<T>(n * n);
	auto d_eigenvalues = cutf::cuda::memory::get_device_unique_ptr<T>(n);
	auto h_matrix_a = cutf::cuda::memory::get_host_unique_ptr<T>(n * n);
	auto h_eigenvalues = cutf::cuda::memory::get_host_unique_ptr<T>(n);

	// print type information{{{
	utils::print_value(test_count, "Test count");
	utils::print_value(std::to_string(n) + " x " + std::to_string(n), "Matrix size");
	utils::print_value(get_type_name<T>(), "Input type");
	utils::print_value(get_type_name<Norm_t>(), "Norm type");
	utils::print_value((UseTC ? "true" : "false"), "Use TC?");
#ifdef PRINT_MATRIX
	utils::print_matrix(a, n, n, "a");
#endif
	// }}}

	// copy
	for(std::size_t i = 0; i < n * n; i++){
		h_matrix_a.get()[i] = cutf::cuda::type::cast<T>(a[i]);
	}
	cutf::cuda::memory::copy(d_matrix_a.get(), h_matrix_a.get(), n * n);
	auto elapsed_time = utils::get_elapsed_time(
			[&d_eigenvalues, &d_matrix_a, &n](){
			for(std::size_t c = 0; c < test_count; c++)
			tcqr::eigen16x16<T, Norm_t, UseTC>(d_eigenvalues.get(), d_matrix_a.get(), n);
			hipDeviceSynchronize();
			});
	utils::print_value(elapsed_time / test_count, "Elapsed time", "ms");

	cutf::cuda::memory::copy(h_eigenvalues.get(), d_eigenvalues.get(), n);
	utils::print_matrix(h_eigenvalues.get(), 1, n, "Eigenvalue");
	std::cout<<std::endl;
}

template void test::time::eigen<half, half, false>(const std::size_t, const float* const);
template void test::time::eigen<half, half, true>(const std::size_t, const float* const);
template void test::time::eigen<half, float, false>(const std::size_t, const float* const);
template void test::time::eigen<half, float, true>(const std::size_t, const float* const);
template void test::time::eigen<float, float, false>(const std::size_t, const float* const);
template void test::time::eigen<float, float, true>(const std::size_t, const float* const);
